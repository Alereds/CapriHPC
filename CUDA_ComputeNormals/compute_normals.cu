#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""
#include "cutil_math.h"
#include <stdio.h>
#include <stdio.h>
#include <iostream>
#include <string>
#include <chrono>
using namespace std;

__device__ float3 PlaneFit(float3 points[10], int nNeighbours)
{
	if (nNeighbours < 3)
		return float3();

	float3 sum = float3();
	for (int i = 0; i < nNeighbours; i++)
		sum += points[i];
	float3 centro = sum * (1.0f / nNeighbours);

	// Calc full 3x3 covariance matrix, excluding symmetries:
	float xx = 0.0f;
	float xy = 0.0f;
	float xz = 0.0f;
	float yy = 0.0f;
	float yz = 0.0f;
	float zz = 0.0f;

	for (int i = 0; i < nNeighbours; i++)
	{
		float3 r = (points[i] - centro);
		xx += r.x * r.x;
		xy += r.x * r.y;
		xz += r.x * r.z;
		yy += r.y * r.y;
		yz += r.y * r.z;
		zz += r.z * r.z;
	}

	float det_x = yy * zz - yz * yz;
	float det_y = xx * zz - xz * xz;
	float det_z = xx * yy - xy * xy;

	float det_max = max(max(det_x, det_y), det_z);

	if (det_max <= 0.0f)
		return float3(); // The points don't span a plane

	// Pick path with best conditioning:
	float3 dir = float3();

	if (det_max == det_x) {
		dir = float3();
		dir.x = det_x;
		dir.y = xz * yz - xy * zz;
		dir.z = xy * yz - xz * yy;
	}
	else if (det_max == det_y) {
		dir = float3();
		dir.x = xz * yz - xy * zz;
		dir.y = det_y;
		dir.z = xy * xz - yz * xx;
	}
	else {
		dir = float3();
		dir.x = xy * yz - xz * yy;
		dir.y = xy * xz - yz * xx;
		dir.z = det_z;
	}

	return normalize(dir);
}

__global__ void ComputeNormals(float3* points, float3* normals, int nRows, int nColumns, int nNearCells, float3 refDirection)
{
	int row = blockIdx.x * blockDim.x + threadIdx.x;
	int column = blockIdx.y * blockDim.y + threadIdx.y;
	if (row >= nRows || column >= nColumns)return;

	int idx = row * nColumns + column; //current index

	int lbr = max(0, row - nNearCells);
	int hbr = min(nRows, row + nNearCells);
	int lbc = max(0, column - nNearCells);
	int hbc = min(nColumns, column + nNearCells);

	float bestDist[10];
	float3 bestPoints[10];
	for (int i = 0; i < 10; i++)
		bestDist[i] = 2e20f;

	int found = 0;

	for (int r = lbr; r < hbr; r++)
	{
		for (int c = lbc; c < hbc; c++)
		{
			// Considering neighbor only if valid and within max distance
			int neighborIdx = r * nColumns + c;

			float dist = distance(points[idx], points[neighborIdx]);
			found++;
			int i = 9;
			while (i > 0 && dist < bestDist[i - 1])
			{
				bestDist[i] = bestDist[i - 1];
				bestPoints[i] = bestPoints[i - 1];
				i--;
			}
			if (i < 9)
			{
				bestDist[i] = dist;
				bestPoints[i] = points[neighborIdx];
			}
		}
	}

	// Compute normals using least squares estimation
	float3 normal = PlaneFit(bestPoints, min(found, 10));

	// Re-orient normal correctly using user reference normals direction
	if (dot(normal, refDirection) > 0.0f)
		normals[idx] = normal;
	else
		normals[idx] = normal * -1; //flip normal
}


int main()
{
	float3* points = NULL;

	string filename = "C:\\Users\\Alessandro\\Desktop\\CUDAComputeNormals\\Gear.pcd";

#pragma region Load from file

	int width, height, nPoints;

	FILE* infile = fopen(filename.c_str(), "r");

	if (infile == NULL) {
		cerr << "Failed to load file!" << endl;
		return;
	}

	char line[100];
	for (int i = 0; i < 6; i++)
		fscanf(infile, "%*[^\n]\n");

	//Width, height
	fscanf(infile, "%*s %i\n", &width);
	fscanf(infile, "%*s %i\n", &height);

	fscanf(infile, "%*[^\n]\n"); //skip one line

	//Points count
	fscanf(infile, "%*s %i\n", &nPoints);

	fscanf(infile, "%*[^\n]\n"); //skip one line

	points = (float3*)malloc(nPoints * sizeof(float3));

	int i = 0;
	while (true)
	{
		float x = 0, y = 0, z = 0, rgb = 0, nx = 0, ny = 0, nz = 0;
		if (fscanf(infile, "%f %f %f %f %f %f %f\n", &x, &y, &z, &rgb, &nx, &ny, &nz) == EOF) { break; }
		points[i].x = x;
		points[i].y = y;
		points[i].z = z;
		i++;
	}

	fclose(infile);

#pragma endregion

	float3* normals = (float3*)malloc(nPoints * sizeof(float3));

	float3* gPoints;
	hipMalloc(&gPoints, nPoints * sizeof(float3));
	float3* gNormals;
	hipMalloc(&gNormals, nPoints * sizeof(float3));

	//Load input points
	hipMemcpy(gPoints, points, nPoints * sizeof(float3), hipMemcpyHostToDevice);

	//Run kernel
	float3 refDirection = float3();
	refDirection.x = 0;
	refDirection.y = 0;
	refDirection.z = -1;

	dim3 dimBlock(32, 32); // so your threads are BLOCK_SIZE*BLOCK_SIZE, 256 in this case
	dim3 dimGrid(ceil(height / 32), ceil(width / 32)); // 1*1 blocks in a grid

	auto start = chrono::steady_clock::now();
	ComputeNormals << <dimGrid, dimBlock >> > (gPoints, gNormals, height, width, 3, refDirection);
	//Get output normals
	hipMemcpy(normals, gNormals, nPoints * sizeof(float3), hipMemcpyDeviceToHost);
	auto end = chrono::steady_clock::now();

	cout << "Elapsed time in milliseconds (CUDA): "
		<< chrono::duration_cast<chrono::milliseconds>(end - start).count()
		<< " ms" << endl;

	filename = "C:\\Users\\Alessandro\\Desktop\\CUDAComputeNormals\\GearWithNormals.pcd";

#pragma region Save to file

	FILE* outfile = fopen(filename.c_str(), "w+");

	string header = "# .PCD v0.7 - Point Cloud Data file format\nVERSION 0.7\nFIELDS x y z normal_x normal_y normal_z\nSIZE 4 4 4 4 4 4\nTYPE F F F F F F\nCOUNT 1 1 1 1 1 1\nWIDTH " + to_string(width) + "\nHEIGHT " + to_string(height) + "\nVIEWPOINT 0 0 0 1 0 0 0\nPOINTS " + to_string(nPoints) + "\nDATA ascii\n";

	if (outfile != NULL)
	{
		fprintf(outfile, "%s", header.c_str());

		for (int i = 0; i < nPoints; i++)
		{
			fprintf(outfile, "%f %f %f %f %f %f\n", points[i].x, points[i].y, points[i].z, normals[i].x, normals[i].y, normals[i].z);
		}
		fclose(outfile);
	}
	else
	{
		cerr << "Unable to save file" << endl;
		perror("Error");
	}

#pragma endregion

	hipFree(gNormals);
	hipFree(gPoints);
	free(normals);
	free(points);
}
